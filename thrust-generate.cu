#include <benchmark/benchmark.h>

#include <thrust/host_vector.h>
#include <thrust/generate.h>
#include <thrust/device_vector.h>
#include <iostream>
#include <hipfft/hipfft.h>
#include <thrust/execution_policy.h>

struct init_int {
	__host__ __device__
	int operator()() {
		return 1;
	}
};

static void thrust_generate_int(benchmark::State& state) {
	int N = state.range(0);
	thrust::host_vector<int> h_vec(N);

	// transfer data to the device
	thrust::device_vector<int> d_vec = h_vec;

	for (auto _ : state) {
		thrust::generate(d_vec.begin(), d_vec.end(), init_int());
	}

	//  Save statistics
	state.SetItemsProcessed(static_cast<int64_t>(state.iterations()) * N);
	state.SetBytesProcessed(
			static_cast<int64_t>(state.iterations()) * N * sizeof(int));
	state.SetComplexityN(N);
}
BENCHMARK(thrust_generate_int)->RangeMultiplier(2)->Range(1<<10, 1<<26)->Complexity();

struct init_cufftComplex {
	__host__ __device__
	hipfftComplex operator()() {
		hipfftComplex s;
		s.x = 0.0f;
		s.y = 0.0f;
		return s;
	}
};

static void thrust_generate_complex(benchmark::State& state) {
	int N = state.range(0);
	thrust::host_vector<hipfftComplex> h_vec(N);

	// transfer data to the device
	thrust::device_vector<hipfftComplex> d_vec = h_vec;

	for (auto _ : state) {
		thrust::generate(d_vec.begin(), d_vec.end(), init_cufftComplex());
	}

	//  Save statistics
	state.SetItemsProcessed(static_cast<int64_t>(state.iterations()) * N);
	state.SetBytesProcessed(
			static_cast<int64_t>(state.iterations()) * N
					* sizeof(hipfftComplex));
	state.SetComplexityN(N);
}
BENCHMARK(thrust_generate_complex)->RangeMultiplier(2)->Range(1<<10, 1<<26)->Complexity();

BENCHMARK_MAIN();
