#include <benchmark/benchmark.h>

#include <thrust/host_vector.h>
#include <thrust/transform.h>
#include <thrust/device_vector.h>
#include <iostream>
#include <hipfft/hipfft.h>
#include <thrust/execution_policy.h>

struct op_increase_int {
  __host__ __device__
  bool operator()(int i) {
    return ++i;
  }
};

static void thrust_transform_int(benchmark::State& state) {
  int N = state.range(0);
  thrust::host_vector<int> h_vec(N);

  // transfer data to the device
  thrust::device_vector<int> d_vec = h_vec;
  thrust::device_vector<int> d_out(N);

  for (auto _ : state) {
    thrust::transform(d_vec.begin(), d_vec.end(), d_out.begin(), op_increase_int());
  }

  //  Save statistics
  state.SetItemsProcessed(static_cast<int64_t>(state.iterations()) * N);
  state.SetBytesProcessed(static_cast<int64_t>(state.iterations()) * N * sizeof(int));
  state.SetComplexityN(N);
}
BENCHMARK(thrust_transform_int)->RangeMultiplier(2)->Range(1<<10, 1<<26)->Complexity();

struct op_increase_complex {
  __host__ __device__
  hipfftComplex operator()(hipfftComplex s) {
    s.x++;
    s.y++;
    return s;
  }
};

static void thrust_transform_complex(benchmark::State& state) {
  int N = state.range(0);
  thrust::host_vector<hipfftComplex> h_vec(N);

  // transfer data to the device
  thrust::device_vector<hipfftComplex> d_vec = h_vec;
  thrust::device_vector<hipfftComplex> d_out(N);

  for (auto _ : state) {
    thrust::transform(d_vec.begin(), d_vec.end(), d_out.begin(), op_increase_complex());
  }

  //  Save statistics
  state.SetItemsProcessed(static_cast<int64_t>(state.iterations()) * N);
  state.SetBytesProcessed(static_cast<int64_t>(state.iterations()) * N * sizeof(hipfftComplex));
  state.SetComplexityN(N);
}
BENCHMARK(thrust_transform_complex)->RangeMultiplier(2)->Range(1<<10, 1<<26)->Complexity();

BENCHMARK_MAIN()
;
